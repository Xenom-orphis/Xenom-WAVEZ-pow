// Optimized Blake3 CUDA implementation for mining
// Based on official Blake3 specification with mining-specific optimizations

#include <hip/hip_runtime.h>
#include <stdint.h>

// Blake3 constants
#define BLAKE3_OUT_LEN 32
#define BLAKE3_BLOCK_LEN 64
#define BLAKE3_CHUNK_LEN 1024

// Blake3 IV (same as SHA-256)
__constant__ uint32_t IV[8] = {
    0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
    0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

// Blake3 message permutation schedule
__constant__ uint8_t MSG_SCHEDULE[7][16] = {
    {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
    {2, 6, 3, 10, 7, 0, 4, 13, 1, 11, 12, 5, 9, 14, 15, 8},
    {3, 4, 10, 12, 13, 2, 7, 14, 6, 5, 9, 0, 11, 15, 8, 1},
    {10, 7, 12, 9, 14, 3, 13, 15, 4, 0, 11, 2, 5, 8, 1, 6},
    {12, 13, 9, 11, 15, 10, 14, 8, 7, 2, 5, 3, 0, 1, 6, 4},
    {9, 14, 11, 5, 8, 12, 15, 1, 13, 3, 0, 10, 2, 6, 4, 7},
    {11, 15, 5, 0, 1, 9, 8, 6, 14, 10, 2, 12, 3, 4, 7, 13},
};

// Blake3 flags
#define CHUNK_START (1 << 0)
#define CHUNK_END (1 << 1)
#define PARENT (1 << 2)
#define ROOT (1 << 3)
#define KEYED_HASH (1 << 4)
#define DERIVE_KEY_CONTEXT (1 << 5)
#define DERIVE_KEY_MATERIAL (1 << 6)

__device__ __forceinline__ uint32_t rotr32(uint32_t w, uint32_t c) {
    return (w >> c) | (w << (32 - c));
}

// Blake3 G function - core mixing function
__device__ __forceinline__ void g(uint32_t *state, uint32_t a, uint32_t b, uint32_t c, uint32_t d,
                                  uint32_t mx, uint32_t my) {
    state[a] = state[a] + state[b] + mx;
    state[d] = rotr32(state[d] ^ state[a], 16);
    state[c] = state[c] + state[d];
    state[b] = rotr32(state[b] ^ state[c], 12);
    state[a] = state[a] + state[b] + my;
    state[d] = rotr32(state[d] ^ state[a], 8);
    state[c] = state[c] + state[d];
    state[b] = rotr32(state[b] ^ state[c], 7);
}

// Blake3 round function - applies G function in column and diagonal pattern
__device__ __forceinline__ void round_fn(uint32_t *state, const uint32_t *msg, const uint8_t *schedule) {
    // Columns
    g(state, 0, 4, 8, 12, msg[schedule[0]], msg[schedule[1]]);
    g(state, 1, 5, 9, 13, msg[schedule[2]], msg[schedule[3]]);
    g(state, 2, 6, 10, 14, msg[schedule[4]], msg[schedule[5]]);
    g(state, 3, 7, 11, 15, msg[schedule[6]], msg[schedule[7]]);
    // Diagonals
    g(state, 0, 5, 10, 15, msg[schedule[8]], msg[schedule[9]]);
    g(state, 1, 6, 11, 12, msg[schedule[10]], msg[schedule[11]]);
    g(state, 2, 7, 8, 13, msg[schedule[12]], msg[schedule[13]]);
    g(state, 3, 4, 9, 14, msg[schedule[14]], msg[schedule[15]]);
}

// Blake3 compression function
__device__ void blake3_compress(const uint32_t cv[8], const uint8_t block[64],
                               uint8_t block_len, uint64_t counter, uint8_t flags,
                               uint32_t out[16]) {
    uint32_t state[16];
    uint32_t block_words[16];
    
    // Initialize state with chaining value and IV
    for (int i = 0; i < 8; i++) {
        state[i] = cv[i];
        state[i + 8] = IV[i];
    }
    
    // Set counter, block length, and flags
    state[12] = (uint32_t)counter;
    state[13] = (uint32_t)(counter >> 32);
    state[14] = (uint32_t)block_len;
    state[15] = (uint32_t)flags;
    
    // Load message words (little-endian)
    for (int i = 0; i < 16; i++) {
        block_words[i] = ((uint32_t)block[i * 4 + 0]) |
                        ((uint32_t)block[i * 4 + 1] << 8) |
                        ((uint32_t)block[i * 4 + 2] << 16) |
                        ((uint32_t)block[i * 4 + 3] << 24);
    }
    
    // 7 rounds of mixing
    for (int i = 0; i < 7; i++) {
        round_fn(state, block_words, MSG_SCHEDULE[i]);
    }
    
    // Finalize: XOR the two halves
    for (int i = 0; i < 8; i++) {
        state[i] ^= state[i + 8];
        state[i + 8] ^= cv[i];
    }
    
    // Copy to output
    for (int i = 0; i < 16; i++) {
        out[i] = state[i];
    }
}

// Optimized Blake3 hash function for mining (handles variable-length inputs)
__device__ void blake3_hash_optimized(const uint8_t *input, uint32_t input_len, uint8_t *output) {
    uint32_t cv[8];
    
    // Initialize chaining value with IV
    for (int i = 0; i < 8; i++) {
        cv[i] = IV[i];
    }
    
    uint32_t offset = 0;
    uint64_t chunk_counter = 0;
    
    // Process input in 64-byte blocks
    while (offset < input_len) {
        uint8_t block[64];
        uint32_t block_len = (input_len - offset < 64) ? (input_len - offset) : 64;
        
        // Copy input to block buffer
        for (uint32_t i = 0; i < block_len; i++) {
            block[i] = input[offset + i];
        }
        // Zero-pad remaining bytes
        for (uint32_t i = block_len; i < 64; i++) {
            block[i] = 0;
        }
        
        // Determine flags
        uint8_t flags = 0;
        if (chunk_counter == 0) flags |= CHUNK_START;
        if (offset + block_len >= input_len) flags |= CHUNK_END | ROOT;
        
        // Compress block
        uint32_t out[16];
        blake3_compress(cv, block, (uint8_t)block_len, chunk_counter, flags, out);
        
        // Update chaining value with first 8 words
        for (int i = 0; i < 8; i++) {
            cv[i] = out[i];
        }
        
        offset += block_len;
        if (offset < input_len) chunk_counter++;
    }
    
    // Extract final hash (little-endian)
    for (int i = 0; i < 8; i++) {
        output[i * 4 + 0] = (uint8_t)(cv[i] & 0xFF);
        output[i * 4 + 1] = (uint8_t)((cv[i] >> 8) & 0xFF);
        output[i * 4 + 2] = (uint8_t)((cv[i] >> 16) & 0xFF);
        output[i * 4 + 3] = (uint8_t)((cv[i] >> 24) & 0xFF);
    }
}

// Fast single-block Blake3 for small inputs (≤64 bytes) - common in mining
__device__ __forceinline__ void blake3_hash_single_block(const uint8_t *input, uint32_t len, uint8_t *output) {
    uint32_t cv[8];
    uint8_t block[64] = {0};
    
    // Initialize CV with IV
    for (int i = 0; i < 8; i++) {
        cv[i] = IV[i];
    }
    
    // Copy input to block
    for (uint32_t i = 0; i < len && i < 64; i++) {
        block[i] = input[i];
    }
    
    // Single compression with appropriate flags
    uint32_t out[16];
    blake3_compress(cv, block, (uint8_t)len, 0, CHUNK_START | CHUNK_END | ROOT, out);
    
    // Extract hash (little-endian)
    for (int i = 0; i < 8; i++) {
        output[i * 4 + 0] = (uint8_t)(out[i] & 0xFF);
        output[i * 4 + 1] = (uint8_t)((out[i] >> 8) & 0xFF);
        output[i * 4 + 2] = (uint8_t)((out[i] >> 16) & 0xFF);
        output[i * 4 + 3] = (uint8_t)((out[i] >> 24) & 0xFF);
    }
}

// Optimized Blake3 batch hashing kernel for mining
extern "C" __global__ void blake3_hash_batch(
    const uint8_t *header_prefix,
    uint32_t header_len,
    const uint8_t *mutation_vectors,
    uint32_t mv_len,
    uint8_t *hashes,
    uint32_t population_size
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= population_size) return;
    
    // Use shared memory for better performance
    __shared__ uint8_t shared_header[256];
    
    // Cooperatively load header to shared memory
    if (threadIdx.x < header_len) {
        shared_header[threadIdx.x] = header_prefix[threadIdx.x];
    }
    __syncthreads();
    
    // Build full input in local memory
    uint8_t buffer[512]; // Increased buffer size for larger inputs
    uint32_t total_len = header_len + mv_len;
    
    // Copy header from shared memory
    for (uint32_t i = 0; i < header_len && i < 512; i++) {
        buffer[i] = (i < 256) ? shared_header[i] : header_prefix[i];
    }
    
    // Copy mutation vector for this thread
    const uint8_t *mv = mutation_vectors + (idx * mv_len);
    for (uint32_t i = 0; i < mv_len && (header_len + i) < 512; i++) {
        buffer[header_len + i] = mv[i];
    }
    
    // Compute Blake3 hash
    uint8_t *hash_out = hashes + (idx * 32);
    
    if (total_len <= 64) {
        // Fast path for small inputs (most mining cases)
        blake3_hash_single_block(buffer, total_len, hash_out);
    } else {
        // General path for larger inputs
        blake3_hash_optimized(buffer, total_len, hash_out);
    }
}

// Optimized fitness evaluation kernel with early termination
extern "C" __global__ void evaluate_fitness(
    const uint8_t *hashes,
    const uint8_t *target_bytes,
    float *fitness,
    uint32_t population_size
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= population_size) return;
    
    const uint8_t *hash = hashes + (idx * 32);
    
    // Load target to shared memory for faster access
    __shared__ uint8_t shared_target[32];
    if (threadIdx.x < 32) {
        shared_target[threadIdx.x] = target_bytes[threadIdx.x];
    }
    __syncthreads();
    
    // Compare hash to target (big-endian comparison)
    bool meets_target = true;
    int first_diff_pos = 32;
    
    for (int k = 0; k < 32; k++) {
        uint8_t h = hash[k];
        uint8_t t = shared_target[k];
        
        if (h < t) {
            // hash < target, solution found
            break;
        } else if (h > t) {
            meets_target = false;
            first_diff_pos = k;
            break;
        }
    }
    
    if (meets_target) {
        // Solution found - use maximum fitness
        fitness[idx] = 999999.0f;
        return;
    }
    
    // Calculate weighted distance for genetic algorithm
    float weighted_dist = 0.0f;
    for (int k = 0; k < 32; k++) {
        int diff = (int)hash[k] - (int)shared_target[k];
        // Weight earlier bytes more heavily (big-endian)
        float weight = (32.0f - k) / 32.0f;
        weighted_dist += diff * weight;
    }
    
    // Inverse fitness with better scaling
    fitness[idx] = 1.0f / (1.0f + fabsf(weighted_dist) / 1000.0f);
}

// Optimized genetic operators kernel for evolutionary mining
extern "C" __global__ void genetic_operators(
    const uint8_t *population_current,
    const float *fitness,
    uint8_t *population_next,
    uint32_t *random_seeds,
    uint32_t population_size,
    uint32_t mv_len,
    float mutation_rate
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= population_size) return;
    
    // Fast LCG random number generator
    uint32_t seed = random_seeds[idx];
    
    auto next_rand = [&seed]() -> uint32_t {
        seed = seed * 1664525u + 1013904223u;
        return seed;
    };
    
    // Tournament selection with size 3 for better selection pressure
    uint32_t best_idx = 0;
    float best_fitness = -1.0f;
    
    for (int tournament = 0; tournament < 3; tournament++) {
        uint32_t candidate = next_rand() % population_size;
        if (fitness[candidate] > best_fitness) {
            best_fitness = fitness[candidate];
            best_idx = candidate;
        }
    }
    
    // Select second parent
    uint32_t parent2_idx = 0;
    float parent2_fitness = -1.0f;
    
    for (int tournament = 0; tournament < 3; tournament++) {
        uint32_t candidate = next_rand() % population_size;
        if (fitness[candidate] > parent2_fitness && candidate != best_idx) {
            parent2_fitness = fitness[candidate];
            parent2_idx = candidate;
        }
    }
    
    const uint8_t *parent1 = population_current + (best_idx * mv_len);
    const uint8_t *parent2 = population_current + (parent2_idx * mv_len);
    uint8_t *child = population_next + (idx * mv_len);
    
    // Uniform crossover (better mixing than single-point)
    for (uint32_t i = 0; i < mv_len; i++) {
        child[i] = (next_rand() & 1) ? parent1[i] : parent2[i];
    }
    
    // Adaptive mutation based on fitness
    float adaptive_rate = mutation_rate;
    if (best_fitness < 0.1f) {
        adaptive_rate *= 2.0f; // Increase mutation when stuck
    }
    
    // Bit-flip mutation
    for (uint32_t i = 0; i < mv_len; i++) {
        if ((next_rand() % 10000) < (uint32_t)(adaptive_rate * 10000)) {
            // Flip a random bit instead of replacing entire byte
            uint32_t bit_pos = next_rand() % 8;
            child[i] ^= (1 << bit_pos);
        }
    }
    
    // Update random seed
    random_seeds[idx] = seed;
}

// Brute-force kernel for systematic nonce search
extern "C" __global__ void blake3_brute_force(
    const uint8_t *header_prefix,
    uint32_t header_len,
    uint64_t start_nonce,
    const uint8_t *target_bytes,
    uint8_t *solution_found,
    uint64_t *solution_nonce,
    uint32_t max_iterations,
    uint8_t *solution_hash
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t nonce = start_nonce + idx;
    
    uint8_t buffer[128];
    uint8_t hash[32];
    
    for (uint32_t iter = 0; iter < max_iterations && !(*solution_found); iter++) {
        // Build input: header + nonce (little-endian)
        for (uint32_t i = 0; i < header_len; i++) {
            buffer[i] = header_prefix[i];
        }
        
        // Append nonce as 8 bytes (little-endian)
        for (int i = 0; i < 8; i++) {
            buffer[header_len + i] = (uint8_t)((nonce >> (i * 8)) & 0xFF);
        }
        
        // Hash the input
        uint32_t total_len = header_len + 8;
        if (total_len <= 64) {
            blake3_hash_single_block(buffer, total_len, hash);
        } else {
            blake3_hash_optimized(buffer, total_len, hash);
        }
        
        // Check if hash meets target (hash <= target)
        // Compare as big-endian: most significant byte first
        bool meets_target = true;
        for (int k = 0; k < 32; k++) {
            if (hash[k] < target_bytes[k]) {
                meets_target = true;  // hash < target, definitely valid
                break;
            } else if (hash[k] > target_bytes[k]) {
                meets_target = false; // hash > target, invalid
                break;
            }
            // If equal, continue to next byte
        }
        // If all bytes equal, hash == target, which is valid (meets_target stays true)
        
        if (meets_target) {
            // Atomic update to prevent race conditions
            if (atomicCAS((unsigned int*)solution_found, 0, 1) == 0) {
                *solution_nonce = nonce;
                // Copy hash to output
                for (int i = 0; i < 32; i++) {
                    solution_hash[i] = hash[i];
                }
            }
            return;
        }
        
        nonce += blockDim.x * gridDim.x; // Stride by total thread count
    }
}
